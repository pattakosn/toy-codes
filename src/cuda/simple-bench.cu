#include "hip/hip_runtime.h"
// https://riptutorial.com/cuda/example/32764/very-simple-cuda-code
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include<cstdio>
#include <cmath>
#include<cstdlib>
#include<iostream>
#include <iomanip>

#include <benchmark/benchmark.h>


using namespace std;
typedef unsigned int uint;


const uint N = 1e6;

__device__ uint Val2[N];

__global__ void set0() {
    uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (index < N)
        Val2[index] = 0;
}


static void init_to_0(benchmark::State &state)
{
	for (auto _ : state) {
		int numThreads = state.range(0);
		uint numBlocks = (uint)ceil(N / (double)numThreads);
		set0 <<< numBlocks, numThreads >>>();
	}
}
BENCHMARK(init_to_0)->RangeMultiplier(2)->Range(32, 16*1024);

BENCHMARK_MAIN();
