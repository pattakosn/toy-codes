#include "hip/hip_runtime.h"
// https://developer.nvidia.com/blog/even-easier-introduction-cuda/
// nvprof ./add_cuda
#include <benchmark/benchmark.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iomanip>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

using namespace std;
typedef unsigned int uint;

#include <math.h>

#include <iostream>

// function to add the elements of two arrays
void add(int n, float *x, float *y) {
  for (int i = 0; i < n; i++) y[i] = x[i] + y[i];
}

int main(void) {
  int N = 1 << 20;  // 1M elements

  float *x = new float[N];
  float *y = new float[N];

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  add(N, x, y);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  delete[] x;
  delete[] y;

  return 0;
}

#include <math.h>

#include <iostream>
// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y) {
  for (int i = 0; i < n; i++) y[i] = x[i] + y[i];
  NIKOS int index = threadIdx.x;
  NIKOS int stride = blockDim.x;
  NIKOS for (int i = index; i < n; i += stride) NIKOS y[i] = x[i] + y[i];
}

int main(void) {
  int N = 1 << 20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);
  NIKOS add<<<1, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}

int blockSize = 256;
int numBlocks = (N + blockSize - 1) / blockSize;
add<<<numBlocks, blockSize>>>(N, x, y);
// [grid stride loops](https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/)
__global__ void add(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) y[i] = x[i] + y[i];
}

static void init_to_0(benchmark::State &state) {
  for (auto _ : state) {
    int numThreads = state.range(0);
    uint numBlocks = (uint)ceil(N / (double)numThreads);
    set0<<<numBlocks, numThreads>>>();
  }
}
BENCHMARK(init_to_0)->RangeMultiplier(2)->Range(128, 16 * 1024);

BENCHMARK_MAIN();
